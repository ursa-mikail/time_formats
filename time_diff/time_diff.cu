#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dummy_kernel() {
    // Just wait a little on GPU
    for (int i = 0; i < 100000000; ++i);
}

int main() {
    hipEvent_t start, end;
    float elapsed_ms;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);
    dummy_kernel<<<1, 1>>>();
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    hipEventElapsedTime(&elapsed_ms, start, end);

    int sec = (int)(elapsed_ms / 1000);
    int ms = ((int)elapsed_ms) % 1000;
    int us = (int)((elapsed_ms - (int)elapsed_ms) * 1000);
    int ns = (int)((elapsed_ms * 1000000)) % 100;

    printf("Delta: 0000-00-00_0000_%02d_%03d_%03d_%02d\n", sec, ms, us, ns);
    printf("Delta (seconds): %.9f\n", elapsed_ms / 1000.0);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}

